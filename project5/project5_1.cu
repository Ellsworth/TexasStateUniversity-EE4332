/******************************************************************************
** Erich Scott Ellsworth, wqi7
** EE4332 – MP#5
** Filename: project1_3.cpp
** Due: 4/18/22
**
** Objective:
** Multiply two random matrices using CUDA
**
*******************************************************************************/

// Includes. Need the CUDA framework to be installed.
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

// Size of the tiles on GPU. DO NOT TOUCH. MICROARCH DEPENDANT.
#define TILE_SIZE 32

#define ARRAY_SIZE 2000        // Size of the array in each direction. row x col
#define RNG_RANGE_MIN -199.99  // Min value for the RNG.
#define RNG_RANGE_MAX 199.99   // Max value for the RNG.

/***********************************************************
 * matrixMultiply: Multiplies two matrices using CUDA.
 *
 * A & B : matrices to multiply.
 * C : matrix of the result. Passed by reference
 *
 * https://en.wikipedia.org/wiki/Matrix_multiplication
/***********************************************************/
__global__ void matrixMultiply(float * A, float * B, float * C)  {

    //tile size arrays - to the GPU's shared memory region
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    // ids of thread	
    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;

    // init to zero 
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((ARRAY_SIZE - 1)/ TILE_SIZE) + 1); k++)  {
        
        // copy data to the tile from matrix A
        if ( (Row < ARRAY_SIZE) && (threadIdx.x + (k*TILE_SIZE)) < ARRAY_SIZE) {
        sA[threadIdx.y][threadIdx.x] = A[(Row*ARRAY_SIZE) + threadIdx.x + (k*TILE_SIZE)];
        }
        
        else  {
        sA[threadIdx.y][threadIdx.x] = 0.0; // if outside the boundaries of the matrix
        }
        
        // Move the data from the tile to matrix B.
        if ( Col < ARRAY_SIZE && (threadIdx.y + k*TILE_SIZE) < ARRAY_SIZE) {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*TILE_SIZE)*ARRAY_SIZE + Col];
        }
        else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
       
        // making sure all threads have arrived before multiplicatoins
        __syncthreads();

        // multiplying elements present in the current tile
        for (int j = 0; j < TILE_SIZE; ++j) {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    
    // Saving final result to Matrix C
    if (Row < ARRAY_SIZE && Col < ARRAY_SIZE) {
           C[Row * ARRAY_SIZE + Col] = Cvalue;
    }
}

/***********************************************************
 * printResult: Prints a matrix.
 *
 * matrix: The matrix to be printed.
/***********************************************************/

void printMatrix(float * matrix) {

    for (int row = 0; row < ARRAY_SIZE; row++) {

        for (int col = 0; col < ARRAY_SIZE; col++) {

            printf("%f  ",*(matrix+(row * ARRAY_SIZE) + col));

        }
        printf("\n");
    }

}

/***********************************************************
 * !!! WARNING !!!
 * 
 * DO NOT USE THIS FUNCTION FOR CRYPTOGRAPHY OR ANY USE REQUIRING VERIFIABLY
 * RANDOM NUMBERS. THE BUILT IN PSUEDO-RNG rand() IN C IS FLAWED AND KNOWN TO 
 * HAVE ERRATA.
 * 
 * For more info read: https://codeforces.com/blog/entry/61587
 * 
 * getRandomNumber: Returns a pseudorandom float.
 *
 * min - the smallest number in the range to generate
 * max - the largest number in the range to generate
/***********************************************************/

float getRandomNumber(float min, float max) {
    return min + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (max - min)));
}

/***********************************************************
 * fillMatrixWithRandom - Fills a matrix with random numbers.
 * 
 * max - how many times to iterate thru the array. eg. 5: fill index 0-4.
 * rng_min - the smallest number in the range to generate
 * rng_max - the largest number in the range to generate
/***********************************************************/

void fillMatrixWithRandom(float rng_min, float rng_max, float * matrix) {
    
    for (int i = 0; i < ARRAY_SIZE * ARRAY_SIZE; i++) {
        matrix[i] = getRandomNumber(rng_min, rng_max);
    }
    
}


int main(int argc, char ** argv) {

    clock_t start = clock();

    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;


    hostA = (float *) malloc(sizeof(float) * ARRAY_SIZE * ARRAY_SIZE);
    hostB = (float *) malloc(sizeof(float) * ARRAY_SIZE * ARRAY_SIZE);
    hostC = (float *) malloc(sizeof(float) * ARRAY_SIZE * ARRAY_SIZE);

    fillMatrixWithRandom(RNG_RANGE_MIN, RNG_RANGE_MAX, hostA);
    fillMatrixWithRandom(RNG_RANGE_MIN, RNG_RANGE_MAX, hostB);
    
    // Allocating GPU memory
    hipMalloc((void **)&deviceA, sizeof(float) *ARRAY_SIZE * ARRAY_SIZE);
    hipMalloc((void **)&deviceB, sizeof(float) *ARRAY_SIZE * ARRAY_SIZE);
    hipMalloc((void **)&deviceC, sizeof(float) *ARRAY_SIZE * ARRAY_SIZE);

    // Copy memory to the GPU
    hipMemcpy(deviceA, hostA, sizeof(float) * ARRAY_SIZE * ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float) * ARRAY_SIZE * ARRAY_SIZE, hipMemcpyHostToDevice);

    // Initialize the grid and block dimensions
    dim3 dimGrid((ARRAY_SIZE/TILE_SIZE) + 1, (ARRAY_SIZE/TILE_SIZE) + 1, 1);	//Number of Blocks required
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1); //Number of threads in each block

    matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC);

    // Another way to sync theads from the GPU
    hipDeviceSynchronize();

    // Copy the results in GPU memory back to the CPU
    hipMemcpy(hostC, deviceC, sizeof(float) * ARRAY_SIZE * ARRAY_SIZE, hipMemcpyDeviceToHost);

    // Print the resulting matrix.
    printMatrix(hostC);

    // Free the GPU memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    
    // Free the Pointer Memory
    free(hostA);
    free(hostB);
    free(hostC);

    // Stop the total runtime clock.
    clock_t stop = clock();
    
    // Print timing data.
    printf("Elapsed: %f seconds\n", (double)(stop - start) / CLOCKS_PER_SEC);

    return 0;
}